#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "doctest.h"
#include "SignalProcessing.h"
#include "dsp.cuh"
#include <iostream>

TEST_CASE("trial")
{
    std::cout << "Hello!" << std::endl;
}

TEST_CASE("Signal Processing")
{
    Signal1D<float> signal(128, {0, 127});
}

TEST_CASE("DSP")
{
    std::cout << "Testing DSP." << std::endl;
    DspGpuImpl1D<float> dsp(1024, 0, 1024);
    dsp.InitInputSignal();

    dsp.DumpInput("input.txt");

    float* filter_d = nullptr;
    int stride = 7;
    hipError_t e = hipMalloc((void**)&filter_d, stride * sizeof(float));

    hiprandGenerator_t gen;
    CURAND_ERROR_HANDLING(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_ERROR_HANDLING(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CURAND_ERROR_HANDLING(hiprandGenerateUniform(gen, filter_d, stride));

    std::unique_ptr<float[]> filter_h = std::make_unique<float[]>(stride);
    CUDA_ERROR_HANDLING(hipMemcpy(filter_h.get(), filter_d, stride * sizeof(float), hipMemcpyDeviceToHost));

    dsp.ApplyFilter(filter_d, stride);

    dsp.DumpOutput("gpu_output.txt");

    dsp.ApplyFilterRef(filter_h.get(), stride);

    dsp.DumpOutput("cpu_output.txt");

    e = hipFree(filter_d);
}


